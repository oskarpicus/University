#include "hip/hip_runtime.h"
#include <iostream>
#include "ppm.h"
#define FILTER_HEIGHT 3
#define FILTER_WIDTH 3
#define BLOCK_SIZE 24

int filter[FILTER_HEIGHT][FILTER_WIDTH] =
{
    0, -1, 0,
    -1, 5, -1,
    0, -1, 0
};

__global__ void applySharpening(int height, int width, int* red, int* green, int* blue, int* filter, int* outRed, int* outGreen, int* outBlue) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < height && j < width) {
        int newRed = 0, newGreen = 0, newBlue = 0;
        int lineKernel = 0, columnKernel = 0;

        for (int k = FILTER_HEIGHT / 2; k >= -FILTER_HEIGHT / 2; k--) {
            for (int l = FILTER_WIDTH / 2; l >= -FILTER_WIDTH / 2; l--) {
                int line, column;

                if (i - k < 0) {
                    line = 0;
                }
                else {
                    if (i - k > height - 1) {
                        line = height - 1;
                    }
                    else {
                        line = i - k;
                    }
                }

                if (j - l < 0) {
                    column = 0;
                }
                else {
                    if (j - l > width - 1) {
                        column = width - 1;
                    }
                    else {
                        column = j - l;
                    }
                }

                newRed += filter[lineKernel * FILTER_WIDTH + columnKernel] * red[line * width + column];
                newGreen += filter[lineKernel * FILTER_WIDTH + columnKernel]* green[line * width + column];
                newBlue += filter[lineKernel * FILTER_WIDTH + columnKernel] * blue[line * width + column];

                columnKernel++;
                if (columnKernel % FILTER_WIDTH == 0) {
                    columnKernel = 0;
                    lineKernel++;
                }
            }
        }

        newRed = newRed > 255 ? 255 : (newRed < 0 ? 0 : newRed);
        newGreen = newGreen > 255 ? 255 : (newGreen < 0 ? 0 : newGreen);
        newBlue = newBlue > 255 ? 255 : (newBlue < 0 ? 0 : newBlue);

        outRed[i * width + j] = newRed;
        outGreen[i * width + j] = newGreen;
        outBlue[i * width + j] = newBlue;
   }
}

int main()
{
    {
        Image image = readImage("nt-P3.ppm");
        int imageSize = image.width * image.height;

        int* h_red = (int*) malloc(sizeof(int) * image.height * image.width);
        int* h_green = (int*) malloc(sizeof(int) * image.height * image.width);
        int* h_blue  = (int*) malloc(sizeof(int) * image.height * image.width);

        flatten(image, h_red, h_green, h_blue);

        int* d_red, * d_green, * d_blue;
        int* d_outRed, * d_outGreen, * d_outBlue;
        int* d_filter;
        hipMalloc((void**)&d_red, sizeof(int) * imageSize);
        hipMalloc((void**)&d_green, sizeof(int) * imageSize);
        hipMalloc((void**)&d_blue, sizeof(int) * imageSize);

        hipMalloc((void**)&d_outRed, sizeof(int) * imageSize);
        hipMalloc((void**)&d_outGreen, sizeof(int) * imageSize);
        hipMalloc((void**)&d_outBlue, sizeof(int) * imageSize);

        hipMalloc((void**)&d_filter, sizeof(int) * FILTER_WIDTH * FILTER_HEIGHT);

        hipMemcpy(d_red, h_red, sizeof(int) * imageSize, hipMemcpyHostToDevice);
        hipMemcpy(d_green, h_green, sizeof(int) * imageSize, hipMemcpyHostToDevice);
        hipMemcpy(d_blue, h_blue, sizeof(int) * imageSize, hipMemcpyHostToDevice);

        hipMemcpy(d_filter, filter, sizeof(int) * FILTER_WIDTH * FILTER_HEIGHT, hipMemcpyHostToDevice);

        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid;
        dimGrid.x = image.width;// + dimBlock.x;
        dimGrid.y = image.height;// + dimBlock.y;

        applySharpening<<<dimGrid, dimBlock>>>(
                image.height,
                image.width,
                d_red,
                d_green,
                d_blue,
                d_filter,
                d_outRed,
                d_outGreen,
                d_outBlue
        );

        hipDeviceSynchronize();

        std::cout << "Cuda status: " << hipGetLastError() << std::endl;

        hipMemcpy(h_red, d_outRed, sizeof(int) * imageSize, hipMemcpyDeviceToHost);
        hipMemcpy(h_green, d_outGreen, sizeof(int) * imageSize, hipMemcpyDeviceToHost);
        hipMemcpy(h_blue, d_outBlue, sizeof(int) * imageSize, hipMemcpyDeviceToHost);

        Image result = deflatten(h_red, h_green, h_blue, image.height, image.width);
        writeImage("result.ppm", result);

        hipFree(d_red);
        hipFree(d_green);
        hipFree(d_blue);
        hipFree(d_outRed);
        hipFree(d_outGreen);
        hipFree(d_outBlue);

        free(h_red);
        free(h_green);
        free(h_blue);

        std::cout << "done\n";
    }
    return 0;
}
